#include "hip/hip_runtime.h"
extern "C" {
    #include "cuda-pgm2asc.cuh"
};


#define TREE_ARRAY_SIZE 1024
#define Nfilt3 6


void rec_generate_tree_d (char *tree, const char *filter, int i, int n)
{
    assert (i >= 0 && i <= 9);
    assert (n < TREE_ARRAY_SIZE);
	
	if (i == 9) {
		if (filter[4] == 0)
			tree[n] = 2;
		else
			tree[n] = 1;
		
		return;
	}
	
	if (n != -1)
		tree[n] = 1;
	if (filter[i] == 0)
		rec_generate_tree_d (tree, filter, i + 1, n * 2 + 2);
	else if (filter[i] == 1)
		rec_generate_tree_d (tree, filter, i + 1, n * 2 + 3);
	else {
		rec_generate_tree_d (tree, filter, i + 1, n * 2 + 2);
		rec_generate_tree_d (tree, filter, i + 1, n * 2 + 3);
	}
}


__device__
int getpixel_d (pix *p, int x, int y, struct job_d *job, char *tree)
{
	if ( x < 0 || y < 0 || x >= p->x || y >= p->y ) 
	  return 255 & ~7;
  
	if (job->tmp.n_run > 0) {
		int pixel_val = pixel_atp (p, x, y) & ~7;
		int n = -1;
	
	#define IS_BLACK(_dx,_dy) !(pixel_atp (p, x + (_dx), y + (_dy)) >> 7)
	#define IS_WHITE(_dx,_dy) (pixel_atp (p, x + (_dx), y + (_dy)) >> 7)
	#define GO_LEFT n = n * 2 + 2
	#define GO_RIGHT n = n * 2 + 3
	#define CHECK_NO_MATCH if (tree[n] == 0) return pixel_val
	
		if (y == 0) {
			n = 13;
		} else {
			if (x == 0 || IS_BLACK (-1, -1)) 
				GO_RIGHT;
			else  
				GO_LEFT;
	
			if (IS_WHITE (0, -1)) 
				GO_LEFT;
			else  
				GO_RIGHT;
			CHECK_NO_MATCH;
	
			if (x + 1 == p->x || IS_BLACK (+1, -1))
				GO_RIGHT;
			else 
				GO_LEFT;
			CHECK_NO_MATCH;
		}
	
	
		if (x == 0 || IS_BLACK (-1, 0)) 
			GO_RIGHT;
		else 
			GO_LEFT;
		CHECK_NO_MATCH;
	
	
		if (IS_WHITE (0, 0))
			GO_LEFT;
		else
			GO_RIGHT;
		CHECK_NO_MATCH;
	
		if (x + 1 == p->x || IS_BLACK (+1, 0)) 
			GO_RIGHT;
		else 
			GO_LEFT;
		CHECK_NO_MATCH;
	
		if (y + 1 == p->y) {
			n = 8 * n + 21;
		} else {
			if (x == 0 || IS_BLACK (-1, +1)) 
				GO_RIGHT;
			else 
				GO_LEFT;
			CHECK_NO_MATCH;
	
			if (IS_WHITE (0, 1)) 
				GO_LEFT;
			else  
				GO_RIGHT;
			CHECK_NO_MATCH;
	
			if (x + 1 == p->x || IS_BLACK (+1, +1)) 
				GO_RIGHT;
			else 
				GO_LEFT;
		}
	
		CHECK_NO_MATCH;
	
		if (tree[n] == 1) {
			return job->cfg.cs;
		} else {
			return 0;
		}

	}

	return (pixel_atp (p,x,y) & ~7);
}


__device__
int distance_d( pix *p1, box_d *box1, pix *p2, box_d *box2, struct job_d *job, char *tree)
{   
	int rc=0,x,y,v1,v2,i1,i2,rgood=0,rbad=0,x1,y1,x2,y2,dx,dy,dx1,dy1,dx2,dy2;
	int cs = job->cfg.cs;

	x1=box1->x0;
	y1=box1->y0;
	x2=box2->x0;
	y2=box2->y0;
  
	dx1=box1->x1 - box1->x0 + 1;
	dx2=box2->x1 - box2->x0 + 1;
	dx=((dx1>dx2)?dx1:dx2);
  
	dy1=box1->y1 - box1->y0 + 1;
	dy2=box2->y1 - box2->y0 + 1;
	dy=((dy1>dy2)?dy1:dy2);
  
	if(abs(dx1-dx2)>1+dx/16 || abs(dy1-dy2)>1+dy/16) return 100;
	// compare relations to baseline and upper line
	if(2*box1->y1>box1->m3+box1->m4 && 2*box2->y1<box2->m3+box2->m4) rbad += 128;
	if(2*box1->y0>box1->m1+box1->m2 && 2*box2->y0<box2->m1+box2->m2) rbad += 128;
	// compare pixels
	for(y = 0; y < dy; y++) {
	  for(x = 0; x < dx; x++) {	// try global shift too ???
		v1 = ((getpixel_d (p1, x1 + x, y1 + y, job, tree) < cs)?1:0);
		i1=8;	// better gray?
		
		v2 = ((getpixel_d (p2, x2 + x, y2 + y, job, tree) < cs)?1:0);
		i2=8;	// better gray?
		
		if(v1 == v2) {
		  rgood += 8;
		  continue;
		} // all things are right!
		// what about different pixel???
		// test overlap of 8 surounding pixels ??? bad if two nb. are bad
		v1=-1;

		for(i1=-1; i1 < 2; i1++) {
		  for(i2=-1; i2 < 2; i2++) {
			if(i1!=0 || i2!=0){
			  if( ((getpixel_d(p1,x1+x+i1*(1+dx/32),y1+y+i2*(1+dy/32), job, tree)<cs)?1:0)
			  !=((getpixel_d(p2,x2+x+i1*(1+dx/32),y2+y+i2*(1+dy/32), job, tree)<cs)?1:0) ) v1++;
			}
		  }
		}

		if (v1 > 0) rbad+=16*v1;
		else rbad++;    
	  }
	}
  
	if(rgood + rbad) rc = (100*rbad+(rgood+rbad-1))/(rgood+rbad);
	else rc = 99;
	
	return rc;
  }


__global__
void deviceFunc (int n, box_d *boxArr, struct job_d *job, pix *pp, struct return_element *returnArr, char *tree)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	if (i < n && j < n) {
		struct box_d *box3 = boxArr + j;
		struct box_d *box2 = boxArr + i;
		int wac = ((box3->num_ac > 0)?box3->wac_0:100);
		int d;

		if (box2->c == UNKNOWN || (box2->num_ac > 0 && box2->wac_0 < 97)) {
			if (box2->y1 - box2->y0 > 4 && box2->x1 - box2->x0 > 1) {

				int *dist = &(returnArr[i].dist);
				int *j_max = &(returnArr[i].j_max);

				if (box3 == box2 || box3->c == UNKNOWN || wac < job->cfg.certainty);
				else if (box2->y1 - box2->y0 < 5 || box2->x1 - box2->x0 < 3);
				else {

					d = distance_d (pp, box2, pp, box3, job, tree);

					atomicMin (dist, d);

					__syncthreads ();

					if (d == *dist) {
						atomicMax (j_max, j);
					}

					__syncthreads ();

					if (j == *j_max) {
						returnArr[i].box2 = *box2;
						returnArr[i].box4 = *box3;
						returnArr[i].bc = box3->c;
					}
				}
			}
		}
	}
}

struct return_element *deviceFuncCall (job_t *job, pix *pp)
{
	const char filt3[Nfilt3][9] = { 
		{0,0,0, 0,0,1, 1,0,0},
		{0,0,0, 1,0,1, 0,0,0},
		{1,0,0, 0,0,1, 0,0,0},
		{1,1,0, 0,1,0, 2,1,1},
		{0,0,1, 0,0,0, 2,1,0},
		{0,1,0, 0,0,0, 1,2,0}
	};
	
	char tree[TREE_ARRAY_SIZE];

	memset (tree, 0, sizeof(tree));

	for (int f = 0; f < Nfilt3; f++) {
		const char * filter = filt3[f];
		rec_generate_tree_d (tree, filter, 0, -1);
	}

	struct timeval stop, start;

	int n = job->res.boxlist.n;

	box_d *boxArr = (box_d *) malloc (n * sizeof (box_d)); 
	struct job_d *jobArr = (struct job_d *) malloc (sizeof (struct job_d));
	pix *pixArr = (pix *) malloc (sizeof (pix));
	struct return_element *returnArr = (struct return_element *) malloc (n * sizeof (struct return_element));

	int i = 0;
	for_each_data (&(job->res.boxlist)) {
		struct box *tmpBox = (struct box *) list_get_current (&(job->res.boxlist));

		boxArr[i] = {
			tmpBox->x0, tmpBox->x1, tmpBox->y0, tmpBox->y1,
			tmpBox->m1, tmpBox->m2, tmpBox->m3, tmpBox->m4,
			tmpBox->wac[0], tmpBox->num_ac, tmpBox->c,
			tmpBox
		};

		i++;
	} end_for_each (&(job->res.boxlist));

	*jobArr = {
		{
			job->tmp.n_run
		},
		{
			job->cfg.cs,
			job->cfg.certainty
		}
	};

	unsigned char *p_d;
	hipMalloc (&p_d, pp->x * pp->y);
	hipMemcpy (p_d, pp->p, pp->x * pp->y, hipMemcpyHostToDevice);
	*pixArr = {
		p_d, pp->x, pp->y, pp->bpp
	};

	for (int i = 0; i < n; i++) {
		returnArr[i].j_max = -1;
		returnArr[i].dist = 1000;
	}

	box_d *boxArr_d;
	struct job_d *jobArr_d;
	pix *pixArr_d;
	struct return_element *returnArr_d;
	char *treeArr_d;

	hipMalloc (&boxArr_d, n * sizeof (box_d));
	hipMalloc (&jobArr_d, sizeof (struct job_d));
	hipMalloc (&pixArr_d, sizeof (pix));
	hipMalloc (&returnArr_d, n * sizeof (struct return_element));
	hipMalloc (&treeArr_d, sizeof (tree));

	hipMemcpy (boxArr_d, boxArr, n * sizeof (box_d), hipMemcpyHostToDevice);
	hipMemcpy (jobArr_d, jobArr, sizeof (struct job_d), hipMemcpyHostToDevice);
	hipMemcpy (pixArr_d, pixArr, sizeof (pix), hipMemcpyHostToDevice);
	hipMemcpy (treeArr_d, tree, sizeof (tree), hipMemcpyHostToDevice);
	hipMemcpy (returnArr_d, returnArr, n * sizeof (struct return_element), hipMemcpyHostToDevice);


    dim3 threadsPerBlock (16, 16);
    dim3 numBlocksTemp (0, 0);

    if ((n / threadsPerBlock.x) * threadsPerBlock.x < n) {
        numBlocksTemp.x = n / threadsPerBlock.x + 1;
    }
    if ((n / threadsPerBlock.y) * threadsPerBlock.y < n) {
        numBlocksTemp.y = n / threadsPerBlock.y + 1;
    }

    dim3 numBlocks (numBlocksTemp.x, numBlocksTemp.y);


	fprintf (stderr, "\n==========\ngpu compute\n");
	fprintf (stderr, "\ngridDim %d, %d\nblockDim %d, %d\n", numBlocks.x, numBlocks.y, threadsPerBlock.y, threadsPerBlock.y);
	gettimeofday (&start, NULL);
	
	deviceFunc <<<numBlocks, threadsPerBlock>>> (n, boxArr_d, jobArr_d, pixArr_d, returnArr_d, treeArr_d);
	hipDeviceSynchronize ();

	gettimeofday (&stop, NULL);
    fprintf (stderr, "\ntook %lu us\n==========\n", (stop.tv_sec - start.tv_sec) * 1000000 + stop.tv_usec - start.tv_usec);


	fprintf (stderr, "\n\n==========\ndevice to host memcpy\n");
	gettimeofday (&start, NULL);

	hipMemcpy (returnArr, returnArr_d, n * sizeof (struct return_element), hipMemcpyDeviceToHost);

	gettimeofday (&stop, NULL);
    fprintf (stderr, "\ntook %lu us\n==========\n", (stop.tv_sec - start.tv_sec) * 1000000 + stop.tv_usec - start.tv_usec);

	return returnArr;
}